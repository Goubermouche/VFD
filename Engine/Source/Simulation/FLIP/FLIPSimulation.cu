#include "FLIPSimulation.cuh"

#include "Simulation/FLIP/FLIPSimulationKernel.cu"
#include "Compute/Utility/CudaKernelUtility.cuh"

#include <glad/glad.h>
#include <cuda_gl_interop.h>

namespace fe {
	extern "C" {
		void FLIPUploadSimulationData(FLIPSimulationData& data)
		{
			COMPUTE_SAFE(hipMemcpyToSymbol(HIP_SYMBOL(c_FLIPDescription), &data, sizeof(FLIPSimulationData)))
		}

		void FLIPUpdateFluidSDF()
		{
			// Grid size
			int threadCount;
			int blockCount;
			ComputeGridSize(1, 256, blockCount, threadCount);
			FLIPTestKernel << < blockCount, threadCount >> > ();
			COMPUTE_SAFE(hipDeviceSynchronize())
		}
	}
}