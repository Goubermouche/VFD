#include "FLIPSimulation.cuh"

#include "Simulation/FLIP/FLIPSimulationKernel.cu"
#include "Compute/Utility/CudaKernelUtility.cuh"

#include <glad/glad.h>
#include <cuda_gl_interop.h>

namespace fe {
	extern "C" {
		void FLIPUploadSimulationData(FLIPSimulationData& data)
		{
			COMPUTE_SAFE(hipMemcpyToSymbol(HIP_SYMBOL(c_FLIPDescription), &data, sizeof(data)))
		}

		void FLIPUploadMAC(MAC& data)
		{
			COMPUTE_SAFE(hipMemcpyToSymbol(HIP_SYMBOL(c_MACTest), &data, sizeof(MAC))) // !
		}

		void FLIPUploadMACVelocities(MACVelocityField& data)
		{
			COMPUTE_SAFE(hipMemcpyToSymbol(HIP_SYMBOL(c_MAC), &data, sizeof(data))) // !
			LOG("velocities uploaded", "FLIP][MAC", ConsoleColor::Cyan);
		}

		void FLIPUpdateFluidSDF()
		{
			// Grid size
			int threadCount;
			int blockCount;
			ComputeGridSize(1, 256, blockCount, threadCount);
			FLIPTestKernel << < blockCount, threadCount >> > ();
			COMPUTE_SAFE(hipDeviceSynchronize())
		}
	}
}