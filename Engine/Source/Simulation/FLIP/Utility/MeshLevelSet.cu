#include "hip/hip_runtime.h"
#include "MeshLevelSet.cuh"
#include "pch.h"

namespace fe {
	__device__ Array3D<float> d_SDFPhi;
	__device__ Array3D<int> d_SDFClosestTriangles;
	__device__ Array3D<float> symbol;

	static __global__ void TestKernel() {
		printf("Test Kernel\n");
	}

	template <typename T>
	__device__ T NewClamp(const T& n, const T& lower, const T& upper) {
		return max(lower, min(n, upper));
	}

	__device__ glm::vec3 NewGridIndexToPosition(int i, int j, int k, float dx) {
		return { i * dx, j * dx, k * dx };
	}

	__device__ float NewPointToSegmentDistance(const glm::vec3& x0, const glm::vec3& x1, const glm::vec3& x2) {
		glm::vec3 dx = x2 - x1;
		float m2 = glm::length2(dx);
		float s12 = glm::dot(x2 - x0, dx) / m2;
		if (s12 < 0) {
			s12 = 0;
		}
		else if (s12 > 1) {
			s12 = 1;
		}

		return glm::length(x0 - (s12 * x1 + (+-s12) * x2));
	}

	__device__ float NewPointToTriangleDistance(const glm::vec3& x0, const glm::vec3& x1, const glm::vec3& x2, const glm::vec3& x3) {
		glm::vec3 x13 = x1 - x3;
		glm::vec3 x23 = x2 - x3;
		glm::vec3 x03 = x0 - x3;

		float m13 = glm::length2(x13);
		float m23 = glm::length2(x23);
		float d = glm::dot(x13, x23);
		float invdet = 1.0f / fmax(m13 * m23 - d * d, 1e-30f);
		float a = glm::dot(x13, x03);
		float b = glm::dot(x23, x03);

		float w23 = invdet * (m23 * a - d * b);
		float w31 = invdet * (m13 * b - d * a);
		float w12 = 1 - w23 - w31;

		if (w23 >= 0 && w31 >= 0 && w12 >= 0) {
			return glm::length(x0 - (w23 * x1 + w31 * x2 + w12 * x3));
		}
		else {
			if (w23 > 0) {
				float d1 = NewPointToSegmentDistance(x0, x1, x2);
				float d2 = NewPointToSegmentDistance(x0, x1, x3);
				return fmin(d1, d2);
			}
			else if (w31 > 0) {
				// this rules out edge 1-3
				float d1 = NewPointToSegmentDistance(x0, x1, x2);
				float d2 = NewPointToSegmentDistance(x0, x2, x3);
				return fmin(d1, d2);
			}
			else {
				// w12 must be >0, ruling out edge 1-2
				float d1 = NewPointToSegmentDistance(x0, x1, x3);
				float d2 = NewPointToSegmentDistance(x0, x2, x3);
				return fmin(d1, d2);
			}
		}
	}

	static __global__ void ComputeExactBandDistanceFieldKernel(int bandWidth, float DX, glm::ivec3 size, Array3D<int>& intersectionCounts, const glm::vec3* vertices, int vertexCount,const glm::ivec3* triangles, int triangleCount) {
		const uint32_t index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

		float invDX = 1.0f / DX;

		glm::ivec3 t = triangles[index];
		glm::vec3 p = vertices[t.x];
		glm::vec3 q = vertices[t.y];
		glm::vec3 r = vertices[t.z];

		float fip = p.x * invDX;
		float fjp = p.y * invDX;
		float fkp = p.z * invDX;

		float fiq = q.x * invDX;
		float fjq = q.y * invDX;
		float fkq = q.z * invDX;

		float fir = r.x * invDX;
		float fjr = r.y * invDX;
		float fkr = r.z * invDX;

		int i0 = NewClamp(int(fmin(fip, fmin(fiq, fir))) - bandWidth, 0, size.x - 1);
		int j0 = NewClamp(int(fmin(fjp, fmin(fjq, fjr))) - bandWidth, 0, size.y - 1);
		int k0 = NewClamp(int(fmin(fkp, fmin(fkq, fkr))) - bandWidth, 0, size.z - 1);

		int i1 = NewClamp(int(fmax(fip, fmax(fiq, fir))) + bandWidth + 1, 0, size.x - 1);
		int j1 = NewClamp(int(fmax(fjp, fmax(fjq, fjr))) + bandWidth + 1, 0, size.y - 1);
		int k1 = NewClamp(int(fmax(fkp, fmax(fkq, fkr))) + bandWidth + 1, 0, size.z - 1);

		for (int k = k0; k <= k1; k++) {
			for (int j = j0; j <= j1; j++) {
				for (int i = i0; i <= i1; i++) {
					glm::vec3 gPos = NewGridIndexToPosition(i, j, k, DX);
					float d = NewPointToTriangleDistance(gPos, p, q, r);
					if (d < d_SDFPhi(i, j, k)) {
						d_SDFPhi.Set(i, j, k, d);
						d_SDFClosestTriangles.Set(i, j, k, index);
					}
				}
			}
		}
	}

	__host__ void MeshLevelSet::CalculateSDFNew(const glm::vec3* vertices, int vertexCount, const glm::ivec3* triangles, int triangleCount, int bandWidth)
	{
		//MeshVertices = vertices;
		//MeshVertexCount = vertexCount;
		//MeshTriangles = triangles;
		//MeshTriangleCount = triangleCount;

		//Array3D<int> intersectionCounts;
	    // intersectionCounts.Init(Phi.Size.x, Phi.Size.y, Phi.Size.z);

		//glm::vec3 size = Phi.Size;

		//Phi.Fill((size.x + size.y + size.z) * DX);
		//ClosestTriangles.Fill(-1);
		//intersectionCounts.Fill(0);

		//// Init memory
		//Array3D<float> PhiDEVICE = Phi.UploadToDevice();
		//Array3D<int> ClosestTrianglesDEVICE = ClosestTriangles.UploadToDevice();
		//intersectionCounts = intersectionCounts.UploadToDevice();

		//COMPUTE_SAFE(hipMemcpyToSymbol(HIP_SYMBOL(d_SDFPhi), &PhiDEVICE, sizeof(Array3D<float>)));
		//COMPUTE_SAFE(hipMemcpyToSymbol(HIP_SYMBOL(d_SDFClosestTriangles), &ClosestTrianglesDEVICE, sizeof(Array3D<int>)));

		//COMPUTE_SAFE(hipMalloc((void**)&MeshVertices, sizeof(float) * 3 * vertexCount));
		//COMPUTE_SAFE(hipMalloc((void**)&MeshTriangles, sizeof(int) * 3 * triangleCount));

		//// Initialize distances near the mesh
		//{
		//	int threadCount;
		//	int blockCount;
		//	ComputeGridSize(MeshTriangleCount, 256, blockCount, threadCount);
		//	ComputeExactBandDistanceFieldKernel <<< threadCount, blockCount >>> (bandWidth, DX, size, intersectionCounts, MeshVertices, MeshVertexCount, MeshTriangles, MeshTriangleCount);
		//	COMPUTE_SAFE(hipDeviceSynchronize());
		//}

		//// Free memory
		//Phi = PhiDEVICE.UploadToHost();
		//ClosestTriangles = ClosestTrianglesDEVICE.UploadToHost();
		//intersectionCounts.Free();

		//COMPUTE_SAFE(hipMemcpyFromSymbol(&Phi, HIP_SYMBOL(d_SDFPhi), sizeof(Phi), 0, hipMemcpyDeviceToHost));
		//COMPUTE_SAFE(hipMemcpyFromSymbol(&ClosestTriangles, HIP_SYMBOL(d_SDFClosestTriangles), sizeof(Array3D<int>)));

		//COMPUTE_SAFE(hipFree((void**)MeshVertices));
		//COMPUTE_SAFE(hipFree((void**)MeshTriangles));

		//ERR(Phi.Get(0));

		glm::vec3 size = Phi.Size;
		Phi.Fill((size.x + size.y + size.z) * DX);

		//auto PhiDevice = Phi.UploadToDevice();
		//COMPUTE_SAFE(hipMemcpyToSymbol(HIP_SYMBOL(d_SDFPhi), &PhiDevice, sizeof(Array3D<float>)));

		//Phi = PhiDevice.UploadToHost();
		//COMPUTE_SAFE(hipMemcpyFromSymbol(&Phi, HIP_SYMBOL(d_SDFPhi), sizeof(Phi), 0, hipMemcpyDeviceToHost));

		// __________________________
		/*Array3D<float>* device;
		Array3D<float>* host;
		device = new Array3D<float>();
		host = new Array3D<float>();

		host->Init(10, 10, 10);
		host->Fill(5.0f);

		COMPUTE_SAFE(hipMalloc(&device, sizeof(Array3D<float>)));
		COMPUTE_SAFE(hipMemcpy(device, host, sizeof(Array3D<float>), hipMemcpyHostToDevice));

		float* host_grid;
		COMPUTE_SAFE(hipMalloc(&host_grid, sizeof(float)));
		COMPUTE_SAFE(hipMemcpy(host_grid, host->Grid, sizeof(float), hipMemcpyHostToDevice));
		COMPUTE_SAFE(hipMemcpy(&(device->Grid), &host_grid, sizeof(void*), hipMemcpyHostToDevice));

		COMPUTE_SAFE(hipMemcpy(host->Grid, host_grid, sizeof(float), hipMemcpyDeviceToHost));
		ERR(host->Grid[0]);*/
		// __________________________

		Array3D<float> host;
		host.Init(10, 10, 10);
		host.Fill(5);
		Array3D<float> device = host;

		COMPUTE_SAFE(hipMalloc((void**)&device.Grid, host.GetSize()));
		COMPUTE_SAFE(hipMemcpy(device.Grid, host.Grid, host.GetSize(), hipMemcpyHostToDevice));
		COMPUTE_SAFE(hipMemcpyToSymbol(HIP_SYMBOL(symbol), &device, sizeof(device)));

		COMPUTE_SAFE(hipMemcpyFromSymbol(&host, HIP_SYMBOL(symbol), sizeof(void*), 0, hipMemcpyDeviceToHost));
		COMPUTE_SAFE(hipMemcpy(host.Grid, device.Grid, host.GetSize(), hipMemcpyDeviceToHost));

		ERR(host.Grid[0]);
	}
}