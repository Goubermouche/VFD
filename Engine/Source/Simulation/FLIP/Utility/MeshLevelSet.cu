#include "hip/hip_runtime.h"
#include "MeshLevelSet.cuh"
#include "pch.h"

#include "Simulation/FLIP/Utility/Grid3D.cuh"

namespace fe {
	__device__ Array3D<float> d_SDFPhi;
	__device__ Array3D<int> d_SDFClosestTriangles;
	__device__ Array3D<int> d_SDFIntersectionCounts;

	// TODO: move to a cuda math file 
	__device__ __host__ float AccurateLength(const glm::vec3 v) {

		return sqrtl(v.x * v.x + v.y * v.y);
	}

	// TODO: move to a cuda math file 
	__device__ __host__ float AccurateLength2(const glm::vec3 v) {
		float l = AccurateLength(v);
		return l * l;
	}

	__device__ __host__ float PointToSegmentDistance(const glm::vec3& x0, const glm::vec3& x1, const glm::vec3& x2) {
		glm::vec3 dx = x2 - x1;
		float m2 = AccurateLength2(dx);
		float s12 = glm::dot(x2 - x0, dx) / m2;
		if (s12 < 0) {
			s12 = 0;
		}
		else if (s12 > 1) {
			s12 = 1;
		}

		return AccurateLength(x0 - (s12 * x1 + (+-s12) * x2));
	}

	__device__ __host__ float PointToTriangleDistance(const glm::vec3& x0, const glm::vec3& x1, const glm::vec3& x2, const glm::vec3& x3) {
		glm::vec3 x13 = x1 - x3;
		glm::vec3 x23 = x2 - x3;
		glm::vec3 x03 = x0 - x3;

		float m13 = AccurateLength2(x13);
		float m23 = AccurateLength2(x23);
		float d = glm::dot(x13, x23);
		float invDet = 1.0f / fmax(m13 * m23 - d * d, 1e-30f);
		float a = glm::dot(x13, x03);
		float b = glm::dot(x23, x03);

		float w23 = invDet * (m23 * a - d * b);
		float w31 = invDet * (m13 * b - d * a);
		float w12 = 1 - w23 - w31;

		if (w23 >= 0.0f && w31 >= 0.0f && w12 >= 0.0f) {
			return AccurateLength(x0 - (w23 * x1 + w31 * x2 + w12 * x3));
		}
		else {
			if (w23 > 0.0f) {
				float d1 = PointToSegmentDistance(x0, x1, x2);
				float d2 = PointToSegmentDistance(x0, x1, x3);
				return min(d1, d2);
			}
			else if (w31 > 0.0f) {
				float d1 = PointToSegmentDistance(x0, x1, x2);
				float d2 = PointToSegmentDistance(x0, x2, x3);
				return min(d1, d2);
			}
			else {
				float d1 = PointToSegmentDistance(x0, x1, x3);
				float d2 = PointToSegmentDistance(x0, x2, x3);
				return min(d1, d2);
			}
		}
	}

	__device__ __host__ int Orientation(float x1, float y1, float x2, float y2, float* twiceSignedArea) {
		*twiceSignedArea = y1 * x2 - x1 * y2;

		if (*twiceSignedArea > 0) {
			return 1;
		}
		else if (*twiceSignedArea < 0) {
			return -1;
		}
		else if (y2 > y1) {
			return 1;
		}
		else if (y2 < y1) {
			return -1;
		}
		else if (x1 > x2) {
			return 1;
		}
		else if (x1 < x2) {
			return -1;
		}
		else {
			return 0;
		}
	}

	__device__ __host__ bool GetBarycentricCoordinates(
		float x0, float y0,
		float x1, float y1, float x2, float y2, float x3, float y3,
		float* a, float* b, float* c
	) {
		x1 -= x0;
		x2 -= x0;
		x3 -= x0;
		y1 -= y0;
		y2 -= y0;
		y3 -= y0;

		float oa;
		int signA = Orientation(x2, y2, x3, y3, &oa);
		if (signA == 0) {
			return false;
		}

		float ob;
		int signB = Orientation(x3, y3, x1, y1, &ob);
		if (signB != signA) {
			return false;
		}

		float oc;
		int signC = Orientation(x1, y1, x2, y2, &oc);
		if (signC != signA) {
			return false;
		}

		float sum = oa + ob + oc;
		float invSum = 1.0f / sum;

		*a = oa * invSum;
		*b = ob * invSum;
		*c = oc * invSum;

		return true;
	}

	__global__ void CalculateExactBandDistanceFieldKernel(int bandWidth, float DX, float invDX, glm::ivec3 size, const glm::vec3* vertices, int vertexCount, const glm::ivec3* triangles, int triangleCount) {
		const int index = blockIdx.x * blockDim.x + threadIdx.x;

		glm::ivec3 t = triangles[index];

		glm::vec3 p = vertices[t.x];
		glm::vec3 q = vertices[t.y];
		glm::vec3 r = vertices[t.z];

		float fip = p.x * invDX;
		float fjp = p.y * invDX;
		float fkp = p.z * invDX;

		float fiq = q.x * invDX;
		float fjq = q.y * invDX;
		float fkq = q.z * invDX;

		float fir = r.x * invDX;
		float fjr = r.y * invDX;
		float fkr = r.z * invDX;

		int i0 = clamp(int(min(fip, min(fiq, fir))) - bandWidth, 0, size.x - 1);
		int j0 = clamp(int(min(fjp, min(fjq, fjr))) - bandWidth, 0, size.y - 1);
		int k0 = clamp(int(min(fkp, min(fkq, fkr))) - bandWidth, 0, size.z - 1);
		int i1 = clamp(int(max(fip, max(fiq, fir))) + bandWidth + 1, 0, size.x - 1);
		int j1 = clamp(int(max(fjp, max(fjq, fjr))) + bandWidth + 1, 0, size.y - 1);
		int k1 = clamp(int(max(fkp, max(fkq, fkr))) + bandWidth + 1, 0, size.z - 1);

		for (int k = k0; k <= k1; k++) {
			for (int j = j0; j <= j1; j++) {
				for (int i = i0; i <= i1; i++) {
					glm::vec3 pos = GridIndexToPosition(i, j, k, DX);
					float d = PointToTriangleDistance(pos, p, q, r);

					if (d < d_SDFPhi(i, j, k)) {
						d_SDFPhi.Set(i, j, k, d);
						d_SDFClosestTriangles.Set(i, j, k, index);
					}
				}
			}
		}

		// Intersection counts
		j0 = clamp((int)ceil(min(fjp, min(fjq, fjr))), 0, size.y - 1);
		k0 = clamp((int)ceil(min(fkp, min(fkq, fkr))), 0, size.z - 1);
		j1 = clamp((int)floor(max(fjp, max(fjq, fjr))), 0, size.y - 1);
		k1 = clamp((int)floor(max(fkp, max(fkq, fkr))), 0, size.z - 1);

		for (int k = k0; k <= k1; k++) {
			for (int j = j0; j <= j1; j++) {
				float a;
				float b;
				float c;

				if (GetBarycentricCoordinates(j, k, fjp, fkp, fjq, fkq, fjr, fkr, &a, &b, &c)) {
					float fi = a * fip + b * fiq + c * fir;
					int interval = int(ceil(fi));
					if (interval < 0) {
						d_SDFIntersectionCounts.AtomicAdd(0, j, k, 1);
					}
					else if (interval < size.x) {
						d_SDFIntersectionCounts.AtomicAdd(interval, j, k, 1);
					}
				}
			}
		}
	}

	__global__ void CalculateDistanceFieldSignsKernel(int sizeX) {
		int k = blockIdx.x * blockDim.x + threadIdx.x;
		int j = blockIdx.y * blockDim.y + threadIdx.y;
		int count = 0;

		for (int i = 0; i < sizeX; i++) {
			count += d_SDFIntersectionCounts(i, j, k);
			if (count % 2 == 1) {
				d_SDFPhi.Set(i, j, k, -d_SDFPhi(i, j, k));
			}
		}
	}

	__host__ void MeshLevelSet::Init(TriangleMesh& mesh, int resolution, float dx, int bandWidth)
	{
		const auto vertices = mesh.GetVertices();
		const auto triangles = mesh.GetTriangles();

		MeshVertexCount = vertices.size();
		MeshTriangleCount = triangles.size();

		MeshVertices = new glm::vec3[MeshVertexCount];
		MeshTriangles = new glm::ivec3[MeshTriangleCount];

		std::copy(vertices.data(), vertices.data() + MeshVertexCount, MeshVertices);
		std::copy(triangles.data(), triangles.data() + MeshTriangleCount, MeshTriangles);

		Size = { resolution, resolution, resolution };
		DX = dx;
		Phi.Init(resolution + 1, resolution + 1, resolution + 1, 0.0f);
		ClosestTriangles.Init(resolution + 1, resolution + 1, resolution + 1, -1);

		glm::ivec3 size = Phi.Size;

		Array3D<int> intersectionCounts;
		intersectionCounts.Init(size.x, size.y, size.z);

		Phi.Fill((size.x + size.y + size.z) * DX);
		ClosestTriangles.Fill(-1);
		intersectionCounts.Fill(0);

		Array3D<float> phiDevice;
		Array3D<int> closestTrianglesDevice;
		Array3D<int> intersectionCountsDevice;

		Phi.UploadToDevice(phiDevice, d_SDFPhi);
		ClosestTriangles.UploadToDevice(closestTrianglesDevice, d_SDFClosestTriangles);
		intersectionCounts.UploadToDevice(intersectionCountsDevice, d_SDFIntersectionCounts);

		glm::vec3* meshVerticesDevice;
		glm::ivec3* meshTrianglesDevice;

		hipMalloc(&meshVerticesDevice, sizeof(glm::vec3) * MeshVertexCount);
		hipMemcpy(meshVerticesDevice, MeshVertices, sizeof(glm::vec3) * MeshVertexCount, hipMemcpyHostToDevice);

		hipMalloc(&meshTrianglesDevice, sizeof(glm::ivec3) * MeshTriangleCount);
		hipMemcpy(meshTrianglesDevice, MeshTriangles, sizeof(glm::ivec3) * MeshTriangleCount, hipMemcpyHostToDevice);

		{
			int blockCount;
			int threadCount;
			ComputeGridSize(MeshTriangleCount, 128, blockCount, threadCount);
			CalculateExactBandDistanceFieldKernel <<< blockCount, threadCount >>> (bandWidth, DX, 1.0f / DX, size, meshVerticesDevice, MeshVertexCount, meshTrianglesDevice, MeshTriangleCount);
			COMPUTE_SAFE(hipDeviceSynchronize());
		}

		{
			dim3 blockCount;
			dim3 threadCount;
			ComputeGridSize({ resolution, resolution }, { 128, 128 }, blockCount, threadCount);
			CalculateDistanceFieldSignsKernel <<< blockCount, threadCount >>> (size.x);
			COMPUTE_SAFE(hipDeviceSynchronize());
		}

		phiDevice.UploadToHost(Phi);
		intersectionCountsDevice.UploadToHost(intersectionCounts);
		closestTrianglesDevice.UploadToHost(ClosestTriangles);

		phiDevice.DeviceFree();
		intersectionCountsDevice.DeviceFree();
		closestTrianglesDevice.DeviceFree();

		delete[] intersectionCounts.Grid;

		COMPUTE_SAFE(hipFree((void**)meshVerticesDevice));
		COMPUTE_SAFE(hipFree((void**)meshTrianglesDevice));
	}
}