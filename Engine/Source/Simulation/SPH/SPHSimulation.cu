#include "pch.h"
#include "SPHSimulation.cuh"

#include "Compute/Utility/RadixSort/RadixSort.cuh";
#include "Simulation/SPH/SPHSimulationKernel.cu"
#include "Compute/Utility/CudaKernelUtility.cuh"

#include <glad/glad.h>
#include <cuda_gl_interop.h>

namespace fe {
	extern "C" {
		void SPHUploadSimulationData(SPHSimulationData& data) {
			COMPUTE_SAFE(hipMemcpyToSymbol(HIP_SYMBOL(c_SPHDescription), &data, sizeof(data)))
		}

		void SPHIntegrate(const unsigned int oldPositionVBO, const unsigned int newPositionVBO, glm::vec4* oldVelocity, glm::vec4* newVelocity, const int particleCount)
		{
			// Grid size
			int threadCount;
			int blockCount;
			ComputeGridSize(particleCount, 256, blockCount, threadCount);

			// Buffer data
			glm::vec4* oldPosition;
			glm::vec4* newPosition;
			COMPUTE_SAFE(cudaGLMapBufferObject((void**)&oldPosition, oldPositionVBO))
			COMPUTE_SAFE(cudaGLMapBufferObject((void**)&newPosition, newPositionVBO))

			// Kernel
			IntegrateKernel << < blockCount, threadCount >> > (newPosition, oldPosition, newVelocity, oldVelocity);
			COMPUTE_CHECK("Kernel execution failed: IntegrateKernel")

			// Unbind buffers
			COMPUTE_SAFE(cudaGLUnmapBufferObject(oldPositionVBO))
			COMPUTE_SAFE(cudaGLUnmapBufferObject(newPositionVBO))

			COMPUTE_SAFE(hipDeviceSynchronize())
		}

		void SPHCalculateHash(const unsigned int positionVBO, glm::uvec2* particleHash, const int particleCount)
		{
			// Grid size
			int threadCount;
			int blockCount;
			ComputeGridSize(particleCount, 512, blockCount, threadCount);

			// Buffer data
			glm::vec4* position;
			COMPUTE_SAFE(cudaGLMapBufferObject((void**)&position, positionVBO))

			// Kernel
			CalculateHashKernel << < blockCount, threadCount >> > (position, particleHash);
			COMPUTE_CHECK("Kernel execution failed: CalculateHashKernel")

			// Unbind buffers
			COMPUTE_SAFE(cudaGLUnmapBufferObject(positionVBO))

			COMPUTE_SAFE(hipDeviceSynchronize())
		}

		void SPHReorder(const unsigned int oldPositionVBO, glm::vec4* oldVelocity, glm::vec4* sortedPosition, glm::vec4* sortedVelocity,
			glm::uvec2* particleHash, unsigned int* cellStart, const unsigned int particleCount, const unsigned int cellCount)
		{
			// Grid size
			int threadCount;
			int blockCount;
			ComputeGridSize(particleCount, 256, blockCount, threadCount);

			// Set all elements of the array to '0xffffffff'
			COMPUTE_SAFE(hipMemset(cellStart, 0xffffffff, cellCount * sizeof(unsigned int)))

			// Buffer data
			glm::vec4* oldPosition;
			COMPUTE_SAFE(cudaGLMapBufferObject((void**)&oldPosition, oldPositionVBO))

			// Texture data
			unsigned int float4MemorySize = particleCount * sizeof(glm::vec4);
			COMPUTE_SAFE(hipBindTexture(0, oldPositionTexture, oldPosition, float4MemorySize))
			COMPUTE_SAFE(hipBindTexture(0, oldVelocityTexture, oldVelocity, float4MemorySize))

			// Kernel
			ReorderKernel << < blockCount, threadCount >> > (particleHash, cellStart, sortedPosition, sortedVelocity);
			COMPUTE_CHECK("Kernel execution failed: ReorderKernel")

			// Unbind textures
			COMPUTE_SAFE(hipUnbindTexture(oldPositionTexture))
			COMPUTE_SAFE(hipUnbindTexture(oldVelocityTexture))

			// Unbind buffers
			COMPUTE_SAFE(cudaGLUnmapBufferObject(oldPositionVBO))

			COMPUTE_SAFE(hipDeviceSynchronize())
		}

		void SPHCollide(const unsigned int positionVBO, glm::vec4* sortedPosition, glm::vec4* sortedVelocity,
			glm::vec4* oldVelocity, glm::vec4* newVelocity, float* pressure, float* density,
			glm::uvec2* particleHash, unsigned int* cellStart, const unsigned int particleCount, const unsigned int cellCount)
		{
			// Grid size
			int threadCount;
			int blockCount;
			ComputeGridSize(particleCount, 64, blockCount, threadCount);

			// Buffer data
			glm::vec4* newPosition;
			COMPUTE_SAFE(cudaGLMapBufferObject((void**)&newPosition, positionVBO))

			// Texture data
			unsigned int float4MemorySize = particleCount * sizeof(glm::vec4);
			unsigned int float1MemorySize = particleCount * sizeof(float);
			COMPUTE_SAFE(hipBindTexture(0, oldPositionTexture, sortedPosition, float4MemorySize))
			COMPUTE_SAFE(hipBindTexture(0, oldVelocityTexture, sortedVelocity, float4MemorySize))
			COMPUTE_SAFE(hipBindTexture(0, pressureTexture, pressure, float1MemorySize))
			COMPUTE_SAFE(hipBindTexture(0, densityTexture, density, float1MemorySize))
			COMPUTE_SAFE(hipBindTexture(0, particleHashTexture, particleHash, particleCount * sizeof(glm::uvec2)))
			COMPUTE_SAFE(hipBindTexture(0, cellStartTexture, cellStart, cellCount * sizeof(unsigned int)))

			// Kernel
			CalculateDensityKernel <<< blockCount, threadCount >>> (pressure, density);
			COMPUTE_CHECK("Kernel execution failed: CalculateDensityKernel")

			COMPUTE_SAFE(hipDeviceSynchronize())

			// Kernel
			CalculateForceKernel << < blockCount, threadCount >> > (newVelocity, particleHash);
			COMPUTE_CHECK("Kernel execution failed: CalculateForceKernel")

			// Unbind buffers
			COMPUTE_SAFE(cudaGLUnmapBufferObject(positionVBO))

			// Unbind textures
			COMPUTE_SAFE(hipUnbindTexture(oldPositionTexture))
			COMPUTE_SAFE(hipUnbindTexture(oldVelocityTexture))
			COMPUTE_SAFE(hipUnbindTexture(pressureTexture))
			COMPUTE_SAFE(hipUnbindTexture(densityTexture))
			COMPUTE_SAFE(hipUnbindTexture(particleHashTexture))
			COMPUTE_SAFE(hipUnbindTexture(cellStartTexture))

			COMPUTE_SAFE(hipDeviceSynchronize())
		}
	}
}