#include "hip/hip_runtime.h"
#ifndef SPH_SIMULATION_KERNEL_CU
#define SPH_SIMULATION_KERNEL_CU

#include "Compute/Utility/CUDA/cutil_math.h"
#include "Simulation/SPH/SPHSimulationParameters.cuh"

namespace fe {
	texture<float4, 1, hipReadModeElementType> oldPositionTexture;
	texture<float4, 1, hipReadModeElementType> oldVelocityTexture;
	texture<uint2, 1, hipReadModeElementType> particleHashTexture;
	texture<uint32_t, 1, hipReadModeElementType> cellStartTexture;
	texture<float, 1, hipReadModeElementType> pressureTexture;
	texture<float, 1, hipReadModeElementType> densityTexture;

	__constant__ SPHSimulationParameters c_SPHDescription;

	static __device__ void CalculateBoundary(glm::vec3& position, glm::vec3& velocity)
	{
		glm::vec3 worldMin = c_SPHDescription.WorldMin;
		const glm::vec3 worldMax = c_SPHDescription.WorldMax;
		glm::vec3 normal;

		const float bounds = c_SPHDescription.BoundsSoftDistance;
		const float stiffness = c_SPHDescription.BoundsStiffness;
		const float damping0 = c_SPHDescription.BoundsDamping;
		const float damping1 = c_SPHDescription.BoundsDampingCritical;
		float acceleration;
		float difference;

#define  EPS	0.00001f // epsilon 
#define  ADD_BOUNDS0()  acceleration = stiffness * difference - damping0 * glm::dot(normal, velocity);  velocity += acceleration * normal * c_SPHDescription.TimeStep;
#define  ADD_BOUNDS1()  acceleration = stiffness * difference - damping1 * glm::dot(normal, velocity);  velocity += acceleration * normal * c_SPHDescription.TimeStep;

		//if (waveMode) {
		//	float sl = 45;
		//	// slope 
		//	difference = bounds - (position.y - worldMin.y) - (position.z - worldMin.z) * sl;

		//	if (difference > EPS) {
		//		normal = { 0, 1 - sl, sl };
		//		ADD_BOUNDS0();
		//	}

		//	float waveRange = 0.15f;
		//	worldMin.x += waveRange * (1.0f + sinf(c_SPHDescription.Time * 2));
		//}

		// Box bounds
		difference = bounds - position.z + worldMin.z;
		if (difference > EPS) { normal = glm::vec3(0, 0, 1); ADD_BOUNDS1(); }
		difference = bounds + position.z - worldMax.z;
		if (difference > EPS) { normal = glm::vec3(0, 0, -1); ADD_BOUNDS1(); }
		difference = bounds - position.x + worldMin.x;
		if (difference > EPS) { normal = glm::vec3(1, 0, 0); ADD_BOUNDS0(); }
		difference = bounds + position.x - worldMax.x;
		if (difference > EPS) { normal = glm::vec3(-1, 0, 0); ADD_BOUNDS0(); }
		difference = bounds - position.y + worldMin.y;
		if (difference > EPS) { normal = glm::vec3(0, 1, 0); ADD_BOUNDS0(); }
		difference = bounds + position.y - worldMax.y;
		if (difference > EPS) { normal = glm::vec3(0, -1, 0); ADD_BOUNDS0(); }
	}

	static __global__ void IntegrateKernel(glm::vec4* newPosition, glm::vec4* oldPosition, glm::vec4* newVelocity, glm::vec4* oldVelocity) {
		const uint32_t index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

		// Load position and velocity into registers
		const glm::vec4 positionFloat4 = oldPosition[index];
		const glm::vec4 velocityFloat4 = oldVelocity[index];
		glm::vec3 position = glm::vec3(positionFloat4.x, positionFloat4.y, positionFloat4.z);
		glm::vec3 velocity = glm::vec3(velocityFloat4.x, velocityFloat4.y, velocityFloat4.z);

		// Calculate boundary conditions
		CalculateBoundary(position, velocity);

		// Add gravity force to velocity
		velocity += c_SPHDescription.Gravity * c_SPHDescription.TimeStep;
		velocity *= c_SPHDescription.GlobalDamping;

		// Update the position of the particle
		position += velocity * c_SPHDescription.TimeStep;

		// Clamp the position to the world boundaries
		const float b = c_SPHDescription.BoundsHardDistance;
		const glm::vec3 worldMin = c_SPHDescription.WorldMin, wmax = c_SPHDescription.WorldMax;
		if (position.x > wmax.x - b) { position.x = wmax.x - b; }
		if (position.x < worldMin.x + b) { position.x = worldMin.x + b; }
		if (position.y > wmax.y - b) { position.y = wmax.y - b; }
		if (position.y < worldMin.y + b) { position.y = worldMin.y + b; }
		if (position.z > wmax.z - b) { position.z = wmax.z - b; }
		if (position.z < worldMin.z + b) { position.z = worldMin.z + b; }

		// Stores the new position and velocity of the particle
		newPosition[index] = glm::vec4(position.x, position.y, position.z, positionFloat4.w);
		newVelocity[index] = glm::vec4(velocity.x, velocity.y, velocity.z, velocityFloat4.w);
	}

	static __device__ int3 CalculateGridPosition(const glm::vec4 position)
	{
		// Convert a world space position into grid coordinates
		int3 gridPosition;
		const glm::vec3 gridPositionFloat3 = (glm::vec3(position.x, position.y, position.z) - c_SPHDescription.WorldMin) / c_SPHDescription.CellSize;
		gridPosition.x = floor(gridPositionFloat3.x);
		gridPosition.y = floor(gridPositionFloat3.y);
		gridPosition.z = floor(gridPositionFloat3.z);
		return gridPosition;
	}

	static __device__ uint32_t CalculateGridHash(int3 gridPosition)
	{
		// Use the particles position and the grid size to calculate a basic and universal hash
		return __mul24(gridPosition.z, c_SPHDescription.GridSizeYX) + __mul24(gridPosition.y, c_SPHDescription.GridSize.x) + gridPosition.x;
	}

	static __global__ void CalculateHashKernel(glm::vec4* position, glm::uvec2* particleHash)
	{
		const uint32_t index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

		// Calculate the grid position of the particle.
		const int3 gridPosition = CalculateGridPosition(position[index]);

		// Calculate the grid hash of the particle
		const uint32_t gridHash = CalculateGridHash(gridPosition);

		// Use the calculated hash to create a key value pair containing the position index
		particleHash[index] = glm::uvec2(gridHash, index);
	}

	static __global__ void ReorderKernel(glm::uvec2* particleHash, uint32_t* cellStart, glm::vec4* sortedPosition, glm::vec4* sortedVelocity)
	{
		const uint32_t index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

		// Load hash value of the current particle into shared memory
		const glm::uvec2 sortedIndex = particleHash[index];
		__shared__ uint32_t sharedHash[257];

		// Account for the previous particle's hash
		sharedHash[threadIdx.x + 1] = sortedIndex.x;
		// If the current particle is not the first particle in the block, then load the hash value of the previous particle into shared memory
		if (index > 0 && threadIdx.x == 0)
		{
			const volatile glm::uvec2 prevData = particleHash[index - 1];
			sharedHash[0] = prevData.x;
		}

		__syncthreads();

		// If the hash values are not equal, then load the index of the current particle into the cellStart array.
		if (index == 0 || sortedIndex.x != sharedHash[threadIdx.x]) {
			cellStart[sortedIndex.x] = index;
		}

		// Use sorted index as index to textures holding position and velocity data
		const float4 sp = tex1Dfetch(oldPositionTexture, sortedIndex.y);
		const float4 sv = tex1Dfetch(oldVelocityTexture, sortedIndex.y);
		sortedPosition[index] = glm::vec4(sp.x, sp.y, sp.z, sp.w);
		sortedVelocity[index] = glm::vec4(sv.x, sv.y, sv.z, sv.w);
	}

	static __device__ float CalculateCellDensity(const int3 gridPosition, uint32_t index, glm::vec4 position)
	{
		float density = 0.0f;

		// Calculate the grid hash for the current particle
		const uint32_t gridHash = CalculateGridHash(gridPosition);
		// Fetch the start index of the cell in the cellStartTexture.
		const uint32_t bucketStart = tex1Dfetch(cellStartTexture, gridHash);
		// If the start index is 0xffffffff, then the cell is empty and the density is set to 0.
		if (bucketStart == 0xffffffff) {
			return density;
		}

		for (uint16_t i = 0; i < c_SPHDescription.MaxParticlesInCellCount; i++)
		{
			const uint32_t indexOther = bucketStart + i;
			// Fetch the hash value of the current cell from the particleHashTexture.
			const uint2 si = tex1Dfetch(particleHashTexture, indexOther);
			const glm::uvec2 sortedIndex = glm::uvec2(si.x, si.y);

			if (sortedIndex.x != gridHash) {
				break;
			}

			if (indexOther != index)
			{
				// Calculate the relative position between the current particle and the other particle.
				const float4 po = tex1Dfetch(oldPositionTexture, indexOther);
				const glm::vec4 positionOther = glm::vec4(po.x, po.y, po.z, po.w);

				const glm::vec4 p = position - positionOther;
				const float r2 = p.x * p.x + p.y * p.y + p.z * p.z;

				if (r2 < c_SPHDescription.SmoothingRadius)
				{
					const float c = c_SPHDescription.SmoothingRadius - r2;
					density += c * c * c;
				}
			}
		}

		return density;
	}

	static __device__ glm::vec3 CalculatePairForce(glm::vec4 relativePosition, glm::vec4 relativeVelocity, float PPAdd, float PPMultiply)
	{
		// Calculate the distance between the two particles
		const glm::vec3 relPos = *(glm::vec3*)&relativePosition.x;
		const glm::vec3 relVel = *(glm::vec3*)&relativeVelocity.x;
		const float r = max(c_SPHDescription.MinDist, length(relPos));

		// If the distance is less than the minimum distance, the force is set to zero
		glm::vec3 force = glm::vec3(0, 0, 0);

		// If the distance is greater than the minimum distance, the force is calculated.
		if (r < c_SPHDescription.Homogeneity)
		{
			// Scale the force by the distance between the two particles.
			const float c = c_SPHDescription.Homogeneity - r;
			const float pressureTerm = c * c_SPHDescription.SpikyKern * PPAdd / r;
			const float velocityTerm = c_SPHDescription.LapKern * c_SPHDescription.Viscosity;

			force = pressureTerm * relPos + velocityTerm * relVel;
			force *= c * PPMultiply;
		}

		return force;
	}

	static __device__ glm::vec3 CalculateCellForce(int3 gridPosition, uint32_t index, glm::vec4 position, glm::vec4 velocity, float currentPressure, float currentDensity)
	{
		glm::vec3 force = glm::vec3(0, 0, 0);

		// Calculate the grid hash for the current particle.
		const uint32_t gridHash = CalculateGridHash(gridPosition);
		// Fetch the start index of the cell from the cellStartTexture.
		const uint32_t bucketStart = tex1Dfetch(cellStartTexture, gridHash);
		// If the start index is 0xffffffff, then the cell is empty and the force is set to 0.
		if (bucketStart == 0xffffffff) {
			return force;
		}

		for (uint16_t i = 0; i < c_SPHDescription.MaxParticlesInCellCount; i++)
		{
			const uint32_t indexOther = bucketStart + i;
			// Fetch the hash value of the current cell from the particleHashTexture.
			const uint2 si = tex1Dfetch(particleHashTexture, indexOther);
			const glm::uvec2 sortedIndex = glm::uvec2(si.x, si.y);

			if (sortedIndex.x != gridHash) {
				break;
			}

			if (indexOther != index)
			{
				// Fetch the position, velocity, pressure and density of the current cell from the oldPositionTexture, oldVelocityTexture, pressureTexture and densityTexture respectively
				const float4 po = tex1Dfetch(oldPositionTexture, indexOther);
				const float4 vo = tex1Dfetch(oldVelocityTexture, indexOther);
				const glm::vec4 positionOther = glm::vec4(po.x, po.y, po.z, po.w);
				const glm::vec4 velocityOther = glm::vec4(vo.x, vo.y, vo.z, vo.w);
				const float pressureOther = tex1Dfetch(pressureTexture, indexOther);
				const float densityOther = tex1Dfetch(densityTexture, indexOther);

				// If the density of the current cell is less than the minimum density, the density of the current cell is set to the minimum density.
				float d12 = min(c_SPHDescription.MinDens, 1.0f / (currentDensity * densityOther));
				// Calculate the pair force between the current and the other cell
				force += CalculatePairForce(position - positionOther, velocityOther - velocity, currentPressure + pressureOther, d12);
			}
		}

		return force;
	}

	static __global__ void CalculateDensityKernel(float* pressure, float* density)
	{
		const uint32_t index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

		// Fetch the position of the particle from the oldPositionTexture.
		const float4 p = tex1Dfetch(oldPositionTexture, index);
		const glm::vec4 position = glm::vec4(p.x, p.y, p.z, p.w);
		// Calculate the grid position of the particle.
		const int3 gridPos = CalculateGridPosition(position);

		float sum = 0.0f;

		// Calculate the density of the particle
		constexpr int16_t s = 1;
		for (int16_t z = -s; z <= s; z++) {
			for (int16_t y = -s; y <= s; y++) {
				for (int16_t x = -s; x <= s; x++) {
					sum += CalculateCellDensity(gridPos + make_int3(x, y, z), index, position);
				}
			}
		}

		// Use common formulas to calculate density and pressure values
		const float newDensity = sum * c_SPHDescription.Poly6Kern * c_SPHDescription.ParticleMass;
		const float newPressure = (newDensity - c_SPHDescription.RestDensity) * c_SPHDescription.Stiffness;

		// Store the new values
		pressure[index] = newPressure;
		density[index] = newDensity;
	}

	static __global__ void CalculateForceKernel(glm::vec4* newVelocity, glm::uvec2* particleHash)
	{
		const uint32_t index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

		// Fetch the position, velocity, pressure and density of the current cell from the oldPositionTexture, oldVelocityTexture, pressureTexture and densityTexture respectively
		const float4 p = tex1Dfetch(oldPositionTexture, index);
		const float4 cv = tex1Dfetch(oldVelocityTexture, index);
		const glm::vec4 position = glm::vec4(p.x, p.y, p.z, p.w);
		const glm::vec4 currentVelocity = glm::vec4(cv.x, cv.y, cv.z, cv.w);
		const float currentPressure = tex1Dfetch(pressureTexture, index);
		const float currentDensity = tex1Dfetch(densityTexture, index);

		// Calculate the grid position of the particle
		const int3 gridPos = CalculateGridPosition(position);

		// Calculate the force that is being exerted onto the particle
		glm::vec3 velocity = glm::vec3(0, 0, 0);
		constexpr  int16_t s = 1;
		for (int16_t z = -s; z <= s; z++) {
			for (int16_t y = -s; y <= s; y++) {
				for (int16_t x = -s; x <= s; x++) {
					velocity += CalculateCellForce(gridPos + make_int3(x, y, z), index, position, currentVelocity,
						currentPressure, currentDensity);
				}
			}
		}

		const volatile uint32_t si = particleHash[index].y;
		velocity *= c_SPHDescription.ParticleMass * c_SPHDescription.TimeStep;

		// Store the new value
		newVelocity[si] = currentVelocity + glm::vec4(velocity.x, velocity.y, velocity.z, 0.0f);
	}
}

#endif // !SPH_SIMULATION_KERNEL_CU