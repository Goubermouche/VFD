#include "hip/hip_runtime.h"
﻿#include "pch.h"

#include "DFSPHImplementation.h"

#include <cuda_gl_interop.h>
#include "Compute/ComputeHelper.h"
#include "Core/Math/Math.h"

#include <thrust/extrema.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <functional>

namespace vfd
{
	DFSPHImplementation::DFSPHImplementation(const GPUDFSPHSimulationDescription& desc)
		 : m_Description(desc)
	{

		//{
		//	const int N = 1000000;
		//	float* h_vec = (float*)malloc(N * sizeof(float));
		//	for (int i = 0; i < N; i++) {
		//		h_vec[i] = i;
		//	}

		//	float* d_vec;
		//	COMPUTE_SAFE(hipMalloc((void**)&d_vec, N * sizeof(float)));
		//	COMPUTE_SAFE(hipMemcpy(d_vec, h_vec, N * sizeof(float), hipMemcpyHostToDevice));

		//	thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(d_vec);

		//	thrust::device_ptr<float> min_ptr;
		//	{
		//		TIME_SCOPE("max")
		//		min_ptr = thrust::max_element(dev_ptr, dev_ptr + N);
		//	}

		//	float min_value = min_ptr[0];
		//	printf("\nMininum value = %f\n", min_value);
		//	printf("Position = %i\n", &min_ptr[0] - &dev_ptr[0]);
		//}

		InitFluidData();

		// Neighborhood search
		m_NeighborhoodSearch = new NeighborhoodSearch(m_Info.SupportRadius);
		m_NeighborhoodSearch->AddPointSet(m_Particles, m_Info.ParticleCount, true, true, true);
	}

	DFSPHImplementation::~DFSPHImplementation()
	{
		delete m_NeighborhoodSearch;
		delete[] m_Particles;
		delete[] m_Particles0;

		COMPUTE_SAFE(hipFree(d_Info))
		COMPUTE_SAFE(cudaGLUnregisterBufferObject(m_VertexBuffer->GetRendererID()))
	}

	void DFSPHImplementation::OnUpdate()
	{
		if (m_Info.ParticleCount == 0) {
			return;
		}

		// Map OpenGL memory to CUDA memory
		DFSPHParticle* particles;
		COMPUTE_SAFE(cudaGLMapBufferObject(reinterpret_cast<void**>(&particles), m_VertexBuffer->GetRendererID()))

		// Sort all particles based on their radius and position
		m_NeighborhoodSearch->FindNeighbors();
		if (m_IterationCount % 500 == 0) {
			PointSet& pointSet = m_NeighborhoodSearch->GetPointSet(0);
			pointSet.SortField(particles);
		}

		// Simulate
		{
			// Clear accelerations
			ClearAccelerationsKernel <<< m_BlockStartsForParticles, m_ThreadsPerBlock >>> (particles, d_Info);
			COMPUTE_SAFE(hipDeviceSynchronize())

			// Update time step size
			CalculateTimeStepSize(thrust::device_pointer_cast(particles));
		
			// Calculate velocities
			CalculateVelocitiesKernel <<< m_BlockStartsForParticles, m_ThreadsPerBlock >>> (particles, d_Info);
			COMPUTE_SAFE(hipDeviceSynchronize())

			// Calculate positions
			CalculatePositionsKernel <<< m_BlockStartsForParticles, m_ThreadsPerBlock >> > (particles, d_Info);
			COMPUTE_SAFE(hipDeviceSynchronize())
		}

		// Unmap OpenGL memory 
		COMPUTE_SAFE(cudaGLUnmapBufferObject(m_VertexBuffer->GetRendererID()))

		// Debug, after the offline solution gets properly implemented this function only needs to be called once
		// after the simulation finishes baking.
		// COMPUTE_SAFE(hipMemcpy(&m_Info, d_Info, sizeof(DFSPHSimulationInfo), hipMemcpyDeviceToHost))

		m_IterationCount++;
	}

	void DFSPHImplementation::Reset()
	{
		// Reset particle positions and velocity
		for (size_t i = 0; i < m_Info.ParticleCount; i++)
		{
			DFSPHParticle particle{};

			// Particle data
			particle.Position = m_Particles0[i].Position;
			particle.Velocity = m_Particles0[i].Velocity;
			particle.Acceleration = { 0.0f, 0.0f, 0.0f };
			particle.Mass = m_Info.Volume * m_Info.Density0;
			particle.Density = m_Info.Density0;
			particle.Kappa = 0.0f;
			particle.KappaVelocity = 0.0f;

			// Viscosity
			particle.ViscosityDifference = { 0.0f, 0.0f, 0.0f };

			// Surface tension
			particle.MonteCarloSurfaceNormals = { 0.0f, 0.0f, 0.0f };
			particle.MonteCarloSurfaceNormalsSmooth = { 0.0f, 0.0f, 0.0f };
			particle.FinalCurvature = 0.0f;
			particle.DeltaFinalCurvature = 0.0f;
			particle.SmoothedCurvature = 0.0f;
			particle.MonteCarloSurfaceCurvature = 0.0f;
			particle.MonteCarloSurfaceCurvatureSmooth = 0.0f;
			particle.ClassifierInput = 0.0f;
			particle.ClassifierOutput = 0.0f;

			m_Particles[i] = particle;
		}

		m_VertexBuffer->SetData(0, m_Info.ParticleCount * sizeof(DFSPHParticle), m_Particles);
		m_VertexBuffer->Unbind();

		// Map OpenGL memory to CUDA memory
		DFSPHParticle* particles;
		COMPUTE_SAFE(cudaGLMapBufferObject(reinterpret_cast<void**>(&particles), m_VertexBuffer->GetRendererID()))

		CalculateMaxVelocityMagnitude(thrust::device_pointer_cast(particles), 0.0f);

		// Unmap OpenGL memory 
		COMPUTE_SAFE(cudaGLUnmapBufferObject(m_VertexBuffer->GetRendererID()))

		m_IterationCount = 0;

		// Reset the time step size
		m_Info.TimeStepSize = m_Description.TimeStepSize;
		COMPUTE_SAFE(hipMemcpy(d_Info, &m_Info, sizeof(DFSPHSimulationInfo), hipMemcpyHostToDevice))
	}

	const Ref<VertexArray>& DFSPHImplementation::GetVertexArray() const
	{
		return m_VertexArray;
	}

	unsigned int DFSPHImplementation::GetParticleCount() const
	{
		return m_Info.ParticleCount;
	}

	float DFSPHImplementation::GetMaxVelocityMagnitude() const
	{
		return m_MaxVelocityMagnitude;
	}

	float DFSPHImplementation::GetTimeStepSize() const
	{
		return m_Info.TimeStepSize;
	}

	void DFSPHImplementation::InitFluidData()
	{
		glm::ivec3 boxSize = { 20, 20, 20 };
		glm::vec3 boxHalfSize = static_cast<glm::vec3>(boxSize - glm::ivec3(1)) / 2.0f;
		unsigned int boxIndex = 0;

		m_Info.ParticleCount = glm::compMul(boxSize);
		m_Info.ParticleRadius = m_Description.ParticleRadius;
		m_Info.ParticleDiameter = 2.0f * m_Info.ParticleRadius;
		m_Info.SupportRadius = 4.0f * m_Info.ParticleRadius;
		m_Info.TimeStepSize = m_Description.TimeStepSize;
		m_Info.Volume = 0.8f * m_Info.ParticleDiameter * m_Info.ParticleDiameter * m_Info.ParticleDiameter;
		m_Info.Density0 = 1000.0f;
		m_Info.WZero = 0.0f;
		m_Info.Gravity = m_Description.Gravity;

		COMPUTE_SAFE(hipMalloc(reinterpret_cast<void**>(&d_Info), sizeof(DFSPHSimulationInfo)))
		COMPUTE_SAFE(hipMemcpy(d_Info, &m_Info, sizeof(DFSPHSimulationInfo), hipMemcpyHostToDevice))

		m_Particles = new DFSPHParticle[m_Info.ParticleCount];
		m_Particles0 = new DFSPHParticle0[m_Info.ParticleCount];

		// Generate a simple box for the purposes of testing 
		for (int x = 0; x < boxSize.x; x++)
		{
			for (int y = 0; y < boxSize.y; y++)
			{
				for (int z = 0; z < boxSize.z; z++)
				{
					DFSPHParticle particle{};

					// Particle data
					particle.Position = {
						(static_cast<float>(x) - boxHalfSize.x) * m_Info.ParticleDiameter,
						(static_cast<float>(y) - boxHalfSize.y) * m_Info.ParticleDiameter,
						(static_cast<float>(z) - boxHalfSize.z) * m_Info.ParticleDiameter
					};

					particle.Velocity = { 0.0f, 0.0f, 0.0f };
					particle.Acceleration = { 0.0f, 0.0f, 0.0f };
					particle.Mass = m_Info.Volume * m_Info.Density0;
					particle.Density = m_Info.Density0;
					particle.Kappa = 0.0f;
					particle.KappaVelocity = 0.0f;

					// Viscosity
					particle.ViscosityDifference = { 0.0f, 0.0f, 0.0f };

					// Surface tension
					particle.MonteCarloSurfaceNormals = { 0.0f, 0.0f, 0.0f };
					particle.MonteCarloSurfaceNormalsSmooth = { 0.0f, 0.0f, 0.0f };
					particle.FinalCurvature = 0.0f;
					particle.DeltaFinalCurvature = 0.0f;
					particle.SmoothedCurvature = 0.0f;
					particle.MonteCarloSurfaceCurvature = 0.0f;
					particle.MonteCarloSurfaceCurvatureSmooth = 0.0f;
					particle.ClassifierInput = 0.0f;
					particle.ClassifierOutput = 0.0f;

					m_Particles[boxIndex] = particle;
					m_Particles0[boxIndex] = DFSPHParticle0{ particle.Position, particle.Velocity };
					boxIndex++;
				}
			}
		}

		unsigned int threadStarts = 0;
		ComputeHelper::GetThreadBlocks(m_Info.ParticleCount, m_ThreadsPerBlock, m_BlockStartsForParticles, threadStarts);

		m_VertexArray = Ref<VertexArray>::Create();
		m_VertexBuffer = Ref<VertexBuffer>::Create(m_Info.ParticleCount * sizeof(DFSPHParticle));
		m_VertexBuffer->SetLayout({
			{ ShaderDataType::Float3, "a_Position"                         },
			{ ShaderDataType::Float3, "a_Velocity"                         },
			{ ShaderDataType::Float3, "a_Acceleration"                     },
			{ ShaderDataType::Float,  "a_Mass"                             },
			{ ShaderDataType::Float,  "a_Density"                          },
			{ ShaderDataType::Float,  "a_Kappa"                            },
			{ ShaderDataType::Float,  "a_KappaVelocity"                    },
			// Viscosity												   
			{ ShaderDataType::Float3, "a_ViscosityDifference"              },
			// Surface tension											   
			{ ShaderDataType::Float3, "a_MonteCarloSurfaceNormals"         },
			{ ShaderDataType::Float3, "a_MonteCarloSurfaceNormalsSmooth"   },
			{ ShaderDataType::Float,  "a_FinalCurvature"                   },
			{ ShaderDataType::Float,  "a_DeltaFinalCurvature"              },
			{ ShaderDataType::Float,  "a_SmoothedCurvature"                },
			{ ShaderDataType::Float,  "a_MonteCarloSurfaceCurvature"       },
			{ ShaderDataType::Float,  "a_MonteCarloSurfaceCurvatureSmooth" },
			{ ShaderDataType::Float,  "a_ClassifierInput"                  },
			{ ShaderDataType::Float,  "a_ClassifierOutput"                 }
		});

		m_VertexArray->AddVertexBuffer(m_VertexBuffer);
		m_VertexBuffer->SetData(0, m_Info.ParticleCount * sizeof(DFSPHParticle), m_Particles);
		m_VertexBuffer->Unbind();

		// Register buffer as a CUDA resource
		COMPUTE_SAFE(cudaGLRegisterBufferObject(m_VertexBuffer->GetRendererID()))
	}

	void DFSPHImplementation::CalculateTimeStepSize(const thrust::device_ptr<DFSPHParticle>& mappedParticles)
	{
		m_MaxVelocityMagnitudeUnaryOperator.TimeStepSize = m_Info.TimeStepSize;
		CalculateMaxVelocityMagnitude(mappedParticles, 0.1f);

		// Use the highest velocity magnitude to approximate the new time step size
		m_Info.TimeStepSize = 0.4f * (m_Info.ParticleDiameter / sqrt(m_MaxVelocityMagnitude));
		m_Info.TimeStepSize = std::min(m_Info.TimeStepSize, m_Description.MaxTimeStepSize);
		m_Info.TimeStepSize = std::max(m_Info.TimeStepSize, m_Description.MinTimeStepSize);

		// Copy the memory new time step back to the device
		COMPUTE_SAFE(hipMemcpy(d_Info, &m_Info, sizeof(DFSPHSimulationInfo), hipMemcpyHostToDevice))
	}
	void DFSPHImplementation::CalculateMaxVelocityMagnitude(const thrust::device_ptr<DFSPHParticle>& mappedParticles, float initialValue)
	{
		m_MaxVelocityMagnitude = thrust::transform_reduce(
			mappedParticles,
			mappedParticles + m_Info.ParticleCount,
			m_MaxVelocityMagnitudeUnaryOperator,
			initialValue,
			thrust::maximum<float>()
		);
	}
}