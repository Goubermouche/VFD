#include "hip/hip_runtime.h"
﻿#include "pch.h"

#include "DFSPHImplementation.h"

#include <cuda_gl_interop.h>
#include "Compute/ComputeHelper.h"
#include "Core/Math/Math.h"

#include <thrust/extrema.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <functional>

namespace vfd
{
	DFSPHImplementation::DFSPHImplementation(const GPUDFSPHSimulationDescription& desc, const std::vector<Ref<RigidBody>>& rigidBodies)
		 : m_Description(desc), m_RigidBodies(rigidBodies)
	{
		InitFluidData();
		InitRigidBodies();

		// Neighborhood search
		m_NeighborhoodSearch = new NeighborhoodSearch(m_Info.SupportRadius);
		m_NeighborhoodSearch->AddPointSet(m_Particles, m_Info.ParticleCount);
	}

	DFSPHImplementation::~DFSPHImplementation()
	{
		delete m_NeighborhoodSearch;
		delete[] m_Particles;
		delete[] m_Particles0;

		COMPUTE_SAFE(hipFree(d_Info))
		COMPUTE_SAFE(cudaGLUnregisterBufferObject(m_VertexBuffer->GetRendererID()))
	}

	void DFSPHImplementation::OnUpdate()
	{
		if (m_Info.ParticleCount == 0) {
			return;
		}

		// Map OpenGL memory to CUDA memory
		DFSPHParticle* particles;
		COMPUTE_SAFE(cudaGLMapBufferObject(reinterpret_cast<void**>(&particles), m_VertexBuffer->GetRendererID()))

		// Sort all particles based on their radius and position
		m_NeighborhoodSearch->FindNeighbors();
		if (m_IterationCount % 500 == 0) {
			PointSet& pointSet = m_NeighborhoodSearch->GetPointSet(0);
			pointSet.SortField(particles);
		}

		// Simulate
		{
			// Compute boundaries
			ComputeVolumeAndBoundaryKernel <<< m_BlockStartsForParticles, m_ThreadsPerBlock >>> (particles, d_Info, d_RigidBodyData);
			COMPUTE_SAFE(hipDeviceSynchronize())

			// Clear accelerations
			ClearAccelerationsKernel <<< m_BlockStartsForParticles, m_ThreadsPerBlock >>> (particles, d_Info);
			COMPUTE_SAFE(hipDeviceSynchronize())

			// Update time step size
			CalculateTimeStepSize(thrust::device_pointer_cast(particles));
		
			// Calculate velocities
			CalculateVelocitiesKernel <<< m_BlockStartsForParticles, m_ThreadsPerBlock >>> (particles, d_Info);
			COMPUTE_SAFE(hipDeviceSynchronize())

			// Calculate positions
			CalculatePositionsKernel <<< m_BlockStartsForParticles, m_ThreadsPerBlock >>> (particles, d_Info);
			COMPUTE_SAFE(hipDeviceSynchronize())
		}

		// Unmap OpenGL memory 
		COMPUTE_SAFE(cudaGLUnmapBufferObject(m_VertexBuffer->GetRendererID()))

		// Debug, after the offline solution gets properly implemented this function only needs to be called once
		// after the simulation finishes baking.
		// COMPUTE_SAFE(hipMemcpy(&m_Info, d_Info, sizeof(DFSPHSimulationInfo), hipMemcpyDeviceToHost))

		m_IterationCount++;
	}

	void DFSPHImplementation::Reset()
	{
		// Reset particle positions and velocity
		for (size_t i = 0; i < m_Info.ParticleCount; i++)
		{
			DFSPHParticle particle{};

			// Particle data
			particle.Position = m_Particles0[i].Position;
			particle.Velocity = m_Particles0[i].Velocity;
			particle.Acceleration = { 0.0f, 0.0f, 0.0f };
			particle.Mass = m_Info.Volume * m_Info.Density0;
			particle.Density = m_Info.Density0;
			particle.Kappa = 0.0f;
			particle.KappaVelocity = 0.0f;

			// Viscosity
			particle.ViscosityDifference = { 0.0f, 0.0f, 0.0f };

			// Surface tension
			particle.MonteCarloSurfaceNormals = { 0.0f, 0.0f, 0.0f };
			particle.MonteCarloSurfaceNormalsSmooth = { 0.0f, 0.0f, 0.0f };
			particle.FinalCurvature = 0.0f;
			particle.DeltaFinalCurvature = 0.0f;
			particle.SmoothedCurvature = 0.0f;
			particle.MonteCarloSurfaceCurvature = 0.0f;
			particle.MonteCarloSurfaceCurvatureSmooth = 0.0f;
			particle.ClassifierInput = 0.0f;
			particle.ClassifierOutput = 0.0f;

			m_Particles[i] = particle;
		}

		m_VertexBuffer->SetData(0, m_Info.ParticleCount * sizeof(DFSPHParticle), m_Particles);
		m_VertexBuffer->Unbind();

		// Map OpenGL memory to CUDA memory
		DFSPHParticle* particles;
		COMPUTE_SAFE(cudaGLMapBufferObject(reinterpret_cast<void**>(&particles), m_VertexBuffer->GetRendererID()))

		CalculateMaxVelocityMagnitude(thrust::device_pointer_cast(particles), 0.0f);

		// Unmap OpenGL memory 
		COMPUTE_SAFE(cudaGLUnmapBufferObject(m_VertexBuffer->GetRendererID()))

		m_IterationCount = 0;

		// Reset the time step size
		m_Info.TimeStepSize = m_Description.TimeStepSize;
		COMPUTE_SAFE(hipMemcpy(d_Info, &m_Info, sizeof(DFSPHSimulationInfo), hipMemcpyHostToDevice))
	}

	const Ref<VertexArray>& DFSPHImplementation::GetVertexArray() const
	{
		return m_VertexArray;
	}

	unsigned int DFSPHImplementation::GetParticleCount() const
	{
		return m_Info.ParticleCount;
	}

	float DFSPHImplementation::GetMaxVelocityMagnitude() const
	{
		return m_MaxVelocityMagnitude;
	}

	float DFSPHImplementation::GetTimeStepSize() const
	{
		return m_Info.TimeStepSize;
	}

	void DFSPHImplementation::InitRigidBodies()
	{
		d_RigidBodyData = m_RigidBodies[0]->GetDeviceData(m_Info.ParticleCount);
	}

	void DFSPHImplementation::InitFluidData()
	{
		const glm::vec3 boxPosition = { 0.0f, 2.0f, 0.0f };
		const glm::uvec3 boxSize = { 20, 20, 20 };

		glm::vec3 boxHalfSize = static_cast<glm::vec3>(boxSize - glm::uvec3(1)) / 2.0f;
		unsigned int boxIndex = 0;

		m_Info.ParticleCount = glm::compMul(boxSize);
		m_Info.ParticleRadius = m_Description.ParticleRadius;
		m_Info.ParticleDiameter = 2.0f * m_Info.ParticleRadius;
		m_Info.SupportRadius = 4.0f * m_Info.ParticleRadius;
		m_Info.TimeStepSize = m_Description.TimeStepSize;
		m_Info.Volume = 0.8f * m_Info.ParticleDiameter * m_Info.ParticleDiameter * m_Info.ParticleDiameter;
		m_Info.Density0 = 1000.0f;
		m_Info.WZero = 0.0f;
		m_Info.Gravity = m_Description.Gravity;
		m_Info.RigidBodyCount = static_cast<unsigned int>(m_RigidBodies.size());

		COMPUTE_SAFE(hipMalloc(reinterpret_cast<void**>(&d_Info), sizeof(DFSPHSimulationInfo)))
		COMPUTE_SAFE(hipMemcpy(d_Info, &m_Info, sizeof(DFSPHSimulationInfo), hipMemcpyHostToDevice))

		m_Particles = new DFSPHParticle[m_Info.ParticleCount];
		m_Particles0 = new DFSPHParticle0[m_Info.ParticleCount];

		// Generate a simple box for the purposes of testing 
		for (unsigned int x = 0u; x < boxSize.x; x++)
		{
			for (unsigned int y = 0u; y < boxSize.y; y++)
			{
				for (unsigned int z = 0u; z < boxSize.z; z++)
				{
					DFSPHParticle particle{};

					// Particle data
					particle.Position = (static_cast<glm::vec3>(glm::uvec3(x, y, z)) - boxHalfSize) * m_Info.ParticleDiameter + boxPosition;
					particle.Velocity = { 0.0f, 0.0f, 0.0f };
					particle.Acceleration = { 0.0f, 0.0f, 0.0f };
					particle.Mass = m_Info.Volume * m_Info.Density0;
					particle.Density = m_Info.Density0;
					particle.Kappa = 0.0f;
					particle.KappaVelocity = 0.0f;

					// Viscosity
					particle.ViscosityDifference = { 0.0f, 0.0f, 0.0f };

					// Surface tension
					particle.MonteCarloSurfaceNormals = { 0.0f, 0.0f, 0.0f };
					particle.MonteCarloSurfaceNormalsSmooth = { 0.0f, 0.0f, 0.0f };
					particle.FinalCurvature = 0.0f;
					particle.DeltaFinalCurvature = 0.0f;
					particle.SmoothedCurvature = 0.0f;
					particle.MonteCarloSurfaceCurvature = 0.0f;
					particle.MonteCarloSurfaceCurvatureSmooth = 0.0f;
					particle.ClassifierInput = 0.0f;
					particle.ClassifierOutput = 0.0f;

					m_Particles[boxIndex] = particle;
					m_Particles0[boxIndex] = DFSPHParticle0{ particle.Position, particle.Velocity };
					boxIndex++;
				}
			}
		}

		unsigned int threadStarts = 0;
		ComputeHelper::GetThreadBlocks(m_Info.ParticleCount, m_ThreadsPerBlock, m_BlockStartsForParticles, threadStarts);

		m_VertexArray = Ref<VertexArray>::Create();
		m_VertexBuffer = Ref<VertexBuffer>::Create(m_Info.ParticleCount * sizeof(DFSPHParticle));
		m_VertexBuffer->SetLayout({
			{ ShaderDataType::Float3, "a_Position"                         }, // Used
			{ ShaderDataType::Float3, "a_Velocity"                         }, // Used
			{ ShaderDataType::Float3, "a_Acceleration"                     }, // Used
			{ ShaderDataType::Float,  "a_Mass"                             }, 
			{ ShaderDataType::Float,  "a_Density"                          }, 
			{ ShaderDataType::Float,  "a_Kappa"                            }, 
			{ ShaderDataType::Float,  "a_KappaVelocity"                    }, 
			// Viscosity												   
			{ ShaderDataType::Float3, "a_ViscosityDifference"              },
			// Surface tension											   
			{ ShaderDataType::Float3, "a_MonteCarloSurfaceNormals"         },
			{ ShaderDataType::Float3, "a_MonteCarloSurfaceNormalsSmooth"   },
			{ ShaderDataType::Float,  "a_FinalCurvature"                   },
			{ ShaderDataType::Float,  "a_DeltaFinalCurvature"              },
			{ ShaderDataType::Float,  "a_SmoothedCurvature"                },
			{ ShaderDataType::Float,  "a_MonteCarloSurfaceCurvature"       },
			{ ShaderDataType::Float,  "a_MonteCarloSurfaceCurvatureSmooth" },
			{ ShaderDataType::Float,  "a_ClassifierInput"                  },
			{ ShaderDataType::Float,  "a_ClassifierOutput"                 }
		});

		m_VertexArray->AddVertexBuffer(m_VertexBuffer);
		m_VertexBuffer->SetData(0, m_Info.ParticleCount * sizeof(DFSPHParticle), m_Particles);
		m_VertexBuffer->Unbind();

		// Register buffer as a CUDA resource
		COMPUTE_SAFE(cudaGLRegisterBufferObject(m_VertexBuffer->GetRendererID()))
	}

	void DFSPHImplementation::CalculateTimeStepSize(const thrust::device_ptr<DFSPHParticle>& mappedParticles)
	{
		m_MaxVelocityMagnitudeUnaryOperator.TimeStepSize = m_Info.TimeStepSize;
		CalculateMaxVelocityMagnitude(mappedParticles, 0.1f);

		// Use the highest velocity magnitude to approximate the new time step size
		m_Info.TimeStepSize = 0.4f * (m_Info.ParticleDiameter / sqrt(m_MaxVelocityMagnitude));
		m_Info.TimeStepSize = std::min(m_Info.TimeStepSize, m_Description.MaxTimeStepSize);
		m_Info.TimeStepSize = std::max(m_Info.TimeStepSize, m_Description.MinTimeStepSize);

		// Copy the memory new time step back to the device
		COMPUTE_SAFE(hipMemcpy(d_Info, &m_Info, sizeof(DFSPHSimulationInfo), hipMemcpyHostToDevice))
	}

	void DFSPHImplementation::CalculateMaxVelocityMagnitude(const thrust::device_ptr<DFSPHParticle>& mappedParticles, float initialValue)
	{
		m_MaxVelocityMagnitude = thrust::transform_reduce(
			mappedParticles,
			mappedParticles + m_Info.ParticleCount,
			m_MaxVelocityMagnitudeUnaryOperator,
			initialValue,
			thrust::maximum<float>()
		);
	}
}