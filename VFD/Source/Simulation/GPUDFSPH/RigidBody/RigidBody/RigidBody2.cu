#include "pch.h"
#include "RigidBody2.cuh"

#include "Compute/ComputeHelper.h"

namespace vfd
{
	RigidBody2::RigidBody2(const RigidBody2Description& desc)
		: m_Description(desc)
	{
		m_Mesh = Ref<TriangleMesh>::Create(desc.SourceMesh);

		glm::vec3 scale;
		for (int i = 0; i < 3; i++)
		{
			scale[i] = glm::length(glm::vec3(desc.Transform[i]));
		}

		m_Rotation = glm::mat3(
			glm::vec3(desc.Transform[0]) / scale[0],
			glm::vec3(desc.Transform[1]) / scale[1],
			glm::vec3(desc.Transform[2]) / scale[2]
		);

		m_DensityMap = DensityMap2("Resources/b.cdm");
	}

	RigidBody2DeviceData* RigidBody2::GetDeviceData(unsigned int particleCount)
	{
		// TEMP
		const std::vector<glm::vec3> boundaryXJ(particleCount);
		const std::vector<float> boundaryVolume(particleCount);
		m_BoundaryXJ = boundaryXJ;
		m_BoundaryVolume = boundaryVolume;

		auto* temp = new RigidBody2DeviceData();
		RigidBody2DeviceData* device;

		temp->Rotation = m_Rotation;
		temp->BoundaryXJ = ComputeHelper::GetPointer(m_BoundaryXJ);
		temp->BoundaryVolume = ComputeHelper::GetPointer(m_BoundaryVolume);
		temp->Map = m_DensityMap.GetDeviceData();

		COMPUTE_SAFE(hipMalloc(reinterpret_cast<void**>(&device), sizeof(RigidBody2DeviceData)))
		COMPUTE_SAFE(hipMemcpy(device, temp, sizeof(RigidBody2DeviceData), hipMemcpyHostToDevice))

		delete temp;
		return device;
	}

}
