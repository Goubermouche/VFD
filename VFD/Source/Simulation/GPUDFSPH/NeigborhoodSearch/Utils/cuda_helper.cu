#include "cuda_helper.h"

namespace vfdcu {
	void CudaHelper::GetThreadBlocks(unsigned int numberOfElements, unsigned int alignment, /*out*/ unsigned int& numberOfThreadBlocks, /*out*/ unsigned int& numberOfThreads)
	{
		numberOfThreads = (numberOfElements / alignment) * alignment;
		numberOfThreadBlocks = (numberOfElements / alignment);
		if (numberOfElements % alignment != 0)
		{
			numberOfThreads += alignment;
			numberOfThreadBlocks++;
		}
	}

	void CudaHelper::MemcpyHostToDevice(void* host, void* device, size_t size)
	{
		hipError_t cudaStatus = hipMemcpy(device, host, size, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			// throw CUDAMemCopyException("hipMemcpy() failed!");
		}
	}

	void CudaHelper::MemcpyDeviceToHost(void* device, void* host, size_t size)
	{
		hipError_t cudaStatus = hipMemcpy(host, device, size, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
		{
			//throw CUDAMemCopyException("hipMemcpy() failed!");
		}
	}
}