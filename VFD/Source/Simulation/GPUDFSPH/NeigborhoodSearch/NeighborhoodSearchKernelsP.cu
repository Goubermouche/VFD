#include "hip/hip_runtime.h"
#include "pch.h"
#include "NeighborhoodSearchKernelsP.cuh"

#define INT16_RANGE 32767
#define UPDATE_REF_OFFSET -32768

__host__ __device__ inline unsigned int Part1By2P(unsigned int x)
{
	x &= 0x000003ff;                  // x = ---- ---- ---- ---- ---- --98 7654 3210
	x = (x ^ (x << 16)) & 0xff0000ff; // x = ---- --98 ---- ---- ---- ---- 7654 3210
	x = (x ^ (x << 8)) & 0x0300f00f;  // x = ---- --98 ---- ---- 7654 ---- ---- 3210
	x = (x ^ (x << 4)) & 0x030c30c3;  // x = ---- --98 ---- 76-- --54 ---- 32-- --10
	x = (x ^ (x << 2)) & 0x09249249;  // x = ---- 9--8 --7- -6-- 5--4 --3- -2-- 1--0
	return x;
}

__host__ __device__ inline unsigned int MortonCode3P(unsigned int x, unsigned int y, unsigned int z)
{
	return (Part1By2P(z) << 2) + (Part1By2P(y) << 1) + Part1By2P(x);
}

__host__ __device__ inline unsigned int CellIndicesToLinearIndexP(const glm::uvec3& cellDimensions, glm::ivec3& xyz)
{
	return xyz.z * cellDimensions.y * cellDimensions.x + xyz.y * cellDimensions.x + xyz.x;
}

inline __device__ unsigned int CellIndexToMortonMetaGridP(const GridInfo& GridInfo, glm::ivec3 gridCell)
{
	glm::ivec3 metaGridCell = glm::ivec3(
		gridCell.x / CUDA_META_GRID_GROUP_SIZE,
		gridCell.y / CUDA_META_GRID_GROUP_SIZE,
		gridCell.z / CUDA_META_GRID_GROUP_SIZE);

	gridCell.x %= CUDA_META_GRID_GROUP_SIZE;
	gridCell.y %= CUDA_META_GRID_GROUP_SIZE;
	gridCell.z %= CUDA_META_GRID_GROUP_SIZE;
	unsigned int metaGridIndex = CellIndicesToLinearIndexP(GridInfo.MetaGridDimension, metaGridCell);

	return metaGridIndex * CUDA_META_GRID_BLOCK_SIZE + MortonCode3P(gridCell.x, gridCell.y, gridCell.z);
}

__global__ void ComputeMinMaxKernelP(const DFSPHParticle* particles, unsigned int particleCount, float m_SearchRadius, glm::ivec3* minCell, glm::ivec3* maxCell)
{
	unsigned int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (particleIndex >= particleCount) {
		return;
	}

	const glm::vec3 particle = particles[particleIndex].Position;
	glm::ivec3 cell;

	cell.x = (int)floor(particle.x / m_SearchRadius);
	cell.y = (int)floor(particle.y / m_SearchRadius);
	cell.z = (int)floor(particle.z / m_SearchRadius);

	atomicMin(&(minCell->x), cell.x);
	atomicMin(&(minCell->y), cell.y);
	atomicMin(&(minCell->z), cell.z);

	atomicMax(&(maxCell->x), cell.x);
	atomicMax(&(maxCell->y), cell.y);
	atomicMax(&(maxCell->z), cell.z);
}

__global__ void InsertParticlesMortonKernelP(const GridInfo gridInfo, const DFSPHParticle* particles, unsigned int* particleCellIndices, unsigned int* cellParticleCounts, unsigned int* sortIndices)
{
	unsigned int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (particleIndex >= gridInfo.ParticleCount)
	{
		return;
	}

	glm::vec3 gridCellF = (particles[particleIndex].Position - gridInfo.GridMin) * gridInfo.GridDelta;
	glm::ivec3 gridCell = glm::ivec3(int(gridCellF.x), int(gridCellF.y), int(gridCellF.z));
	unsigned int cellIndex = CellIndexToMortonMetaGridP(gridInfo, gridCell);
	particleCellIndices[particleIndex] = cellIndex;
	sortIndices[particleIndex] = atomicAdd(&cellParticleCounts[cellIndex], 1);
}

__global__ void CountingSortIndicesKernelP(const GridInfo gridInfo, const unsigned int* particleCellIndices, const unsigned int* cellOffsets, const unsigned int* sortIndicesSrc, unsigned int* sortIndicesDest)
{
	unsigned int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (particleIndex >= gridInfo.ParticleCount)
	{
		return;
	}

	unsigned int gridCellIndex = particleCellIndices[particleIndex];
	unsigned int sortIndex = sortIndicesSrc[particleIndex] + cellOffsets[gridCellIndex];
	sortIndicesDest[sortIndex] = particleIndex;
}

__global__ void ComputeCountsKernelP(const DFSPHParticle* queryPoints, const unsigned int queryPointCount, const GridInfo gridInfo, const DFSPHParticle* particles, const unsigned int* cellOffsets, const unsigned int* cellParticleCounts, unsigned int* neighborCounts, const unsigned int* reversedSortIndices)
{
	unsigned int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (particleIndex >= queryPointCount)
	{
		return;
	}

	const glm::vec3 particle = queryPoints[particleIndex].Position;
	glm::vec3 gridCellF = (particle - gridInfo.GridMin) * gridInfo.GridDelta;
	glm::ivec3 coord = glm::ivec3(int(floor(gridCellF.x)), int(floor(gridCellF.y)), int(floor(gridCellF.z)));
	unsigned int neighborCount = 0;

	for (int z = -1; z < 2; z++)
	{
		for (int y = -1; y < 2; y++)
		{
			for (int x = -1; x < 2; x++)
			{
				glm::ivec3 finalCoord = coord + glm::ivec3(x, y, z);

				if (finalCoord.x < 0 || finalCoord.y < 0 || finalCoord.z < 0
					|| finalCoord.x >= gridInfo.GridDimension.x || finalCoord.y >= gridInfo.GridDimension.y || finalCoord.z >= gridInfo.GridDimension.z)
				{
					continue;
				}

				unsigned int neighborCellIndex = CellIndexToMortonMetaGridP(gridInfo, finalCoord);
				unsigned int neighborCellCount = cellParticleCounts[neighborCellIndex];
				unsigned int neighborCellStart = cellOffsets[neighborCellIndex];

				for (unsigned int i = neighborCellStart; i < neighborCellStart + neighborCellCount; i++)
				{
					unsigned int& neighborIndex = i;
					glm::vec3 diff = particles[reversedSortIndices[neighborIndex]].Position - particle;
					float squaredDistance = diff.x * diff.x + diff.y * diff.y + diff.z * diff.z;

					if (squaredDistance < gridInfo.SquaredSearchRadius && squaredDistance > 0.0)
					{
						neighborCount++;
					}

					if (neighborCount == CUDA_MAX_NEIGHBORS)
					{
						neighborCounts[particleIndex] = neighborCount;
						return;
					}
				}
			}
		}
	}

	neighborCounts[particleIndex] = neighborCount;
}

__global__ void NeighborhoodQueryWithCountsKernelP(const DFSPHParticle* queryPoints, const unsigned int queryPointCount, const GridInfo gridInfo, const DFSPHParticle* particles, const unsigned int* cellOffsets, const unsigned int* cellParticleCounts, const unsigned int* neighborWriteOffsets, unsigned int* neighbors, const unsigned int* reversedSortIndices)
{
	unsigned int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (particleIndex >= queryPointCount)
	{
		return;
	}

	const glm::vec3 particle = queryPoints[particleIndex].Position;
	glm::vec3 gridCellF = (particle - gridInfo.GridMin) * gridInfo.GridDelta;
	glm::ivec3 coord = glm::ivec3(int(floor(gridCellF.x)), int(floor(gridCellF.y)), int(floor(gridCellF.z)));
	unsigned int neighborCount = 0;
	const unsigned int writeOffset = neighborWriteOffsets[particleIndex];

	for (int z = -1; z < 2; z++)
	{
		for (int y = -1; y < 2; y++)
		{
			for (int x = -1; x < 2; x++)
			{
				glm::ivec3 finalCoord = coord + glm::ivec3(x, y, z);

				if (finalCoord.x < 0 || finalCoord.y < 0 || finalCoord.z < 0
					|| finalCoord.x >= gridInfo.GridDimension.x || finalCoord.y >= gridInfo.GridDimension.y || finalCoord.z >= gridInfo.GridDimension.z)
				{
					continue;
				}

				unsigned int neighborCellIndex = CellIndexToMortonMetaGridP(gridInfo, finalCoord);
				unsigned int neighborCellCount = cellParticleCounts[neighborCellIndex];
				unsigned int neighborCellStart = cellOffsets[neighborCellIndex];

				for (unsigned int i = neighborCellStart; i < neighborCellStart + neighborCellCount; i++)
				{
					unsigned int& neighborIndex = i;
					glm::vec3 diff = particles[reversedSortIndices[neighborIndex]].Position - particle;
					float squaredDistance = diff.x * diff.x + diff.y * diff.y + diff.z * diff.z;

					if (squaredDistance < gridInfo.SquaredSearchRadius && squaredDistance > 0.0)
					{
						neighbors[writeOffset + neighborCount] = reversedSortIndices[neighborIndex];
						neighborCount++;
					}

					if (neighborCount == CUDA_MAX_NEIGHBORS)
					{
						return;
					}
				}
			}
		}
	}
}